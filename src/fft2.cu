#include "hip/hip_runtime.h"
/*     fft2.cu		                                                         */
/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
/*                                                                               */
/* Copyright (C) 2015                                                            */
/*                                                                               */
/*       <eInfochips Ltd.>                                                       */
/*                                                                               */
/*    This library is free software; you can redistribute it and/or              */
/*    modify it under the terms of the GNU Lesser General Public                 */
/*    License as published by the Free Software Foundation; either               */
/*    version 2.1 of the License, or (at your option) any later version.         */
/*                                                                               */
/*    This library is distributed in the hope that it will be useful,            */
/*    but WITHOUT ANY WARRANTY; without even the implied warranty of             */
/*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU          */
/*    Lesser General Public License for more details.                            */
/*                                                                               */
/*    You should have received a copy of the GNU Lesser General Public           */
/*    License along with this library; if not, write to the Free Software        */
/*    Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA  */
/*                                                                               */
/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
/*                                                                               */
/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/
/* Written by:                                                                   */
/*      Pratik Bari                                                              */
/*      pratik.bari@einfochips.com                                               */
/*~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~*/

#include "fft2.cuh"
#include "common.h"

/**************************************************** Function for the Memory Free **********************************************************/

void free_fft(double *in, hipfftDoubleComplex *out) {

	if(in != NULL) {
		free(in);
		in = NULL;
	}

	if(out != NULL) {
		free(out);
		out = NULL;
	}

	hipDeviceReset();
}

extern "C" void
gpu_fft2 (ImageMatrix *obj, const ImageMatrix &matrix_IN) {

	hipfftHandle plan;
	double *in = NULL;
	hipfftDoubleComplex *out = NULL;
	hipfftDoubleReal *d_in = NULL;
	hipfftDoubleComplex *d_out = NULL;

	unsigned int half_height = matrix_IN.height/2+1;
	unsigned int height = matrix_IN.height;
	unsigned int width = matrix_IN.width;

	obj->copyFields(matrix_IN);
	obj->allocate (matrix_IN.width, matrix_IN.height);
	writeablePixels out_plane = obj->WriteablePixels();
	readOnlyPixels in_plane = matrix_IN.ReadablePixels();

/********************************************************* Host Memory Allocations ************************************************************/

	in = (double*)malloc(sizeof(double) * width * height);	 // Host Memory for input
	if(in == NULL){
		printf("Error in malloc : 'in' in fft2");
		free_fft(in, out);
		exit(1);
	}

	out = (hipfftDoubleComplex *)malloc(sizeof(hipfftDoubleComplex) * width * height); // Host memory for output
	if(out == NULL){
		printf("Error in malloc : 'out' in fft2");
		free_fft(in, out);
		exit(1);
	}

/******************************************************* Device Memory Allocations *************************************************************/

	if(checkCudaErrors(hipMalloc((void **)&d_in, sizeof(hipfftDoubleReal) * width * height)) != hipSuccess) { // Device Memory for input
		free_fft(in, out);
		exit(1);
	}

	if(checkCudaErrors(hipMalloc((void **)&d_out, sizeof(hipfftDoubleComplex) * width * height)) != hipSuccess) {	// Device Memory for output
		free_fft(in, out);
		exit(1);
	}

	unsigned int x,y;
	for (x=0;x<width;x++){
		for (y=0;y<height;y++)
			in[height*x+y]=in_plane.coeff(y,x); // Initialization of the data in host input
	}

/***************************************************** Host To Device Memcpy operations ***********************************************************/

	if(checkCudaErrors(hipMemcpy(d_in, in, sizeof(double) * width * height, hipMemcpyHostToDevice)) != hipSuccess) { // Input Memcpy
		free_fft(in, out);
		exit(1);
	}

/*************************************************************** Plan Creation *********************************************************************/

	if (checkCudaErrors(hipfftPlan2d(&plan, width, height, HIPFFT_D2Z)) != HIPFFT_SUCCESS){
		free_fft(in, out);
		exit(1);
	}

/************************************************************ Execution of the Plan ***************************************************************/

	if (checkCudaErrors(hipfftExecD2Z(plan, d_in, d_out)) != HIPFFT_SUCCESS){
		free_fft(in, out);
		exit(1);
	}

/***************************************************** Device To Host Memcpy operations **********************************************************/

	if(checkCudaErrors(hipMemcpy(out, d_out, sizeof(hipfftDoubleComplex) * width * height, hipMemcpyDeviceToHost)) != hipSuccess) {
		free_fft(in, out);
		exit(1);
	}

	unsigned long idx;
	for (x=0;x<width;x++) {
		for (y=0;y<half_height;y++) {
			idx = half_height*x+y;
			double ri = out[idx].x;
			double im = out[idx].y;
			out_plane (y,x) = obj->stats.add (sqrt((ri * ri)+(im * im))); // sqrt(real(X).^2 + imag(X).^2)
		}
	}

	// complete the first column
	for (y=half_height;y<height;y++)
		out_plane (y,0) = obj->stats.add (out_plane (height - y, 0));

	// complete the rest of the columns
	for (y=half_height;y<height;y++)
		for (x=1;x<width;x++)   // 1 because the first column is already completed
			out_plane (y,x) =obj-> stats.add (out_plane (height - y, width - x));

	// clean up
	if(checkCudaErrors(hipfftDestroy(plan)) != HIPFFT_SUCCESS) {
		free_fft(in, out);
		exit(1);
	}

	if(checkCudaErrors(hipFree(d_in)) != hipSuccess) {
		free_fft(in, out);
		exit(1);
	}

	if(checkCudaErrors(hipFree(d_out)) != hipSuccess) {
		free_fft(in, out);
		exit(1);
	}

	free(in);
	free(out);
}
